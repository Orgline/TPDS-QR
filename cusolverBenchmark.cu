#include "TensorBLAS.h"

int m = 0, n = 0, type = 0;
float ms = 0;

int parseArguments(int argc,char *argv[])
{
    if(argc < 4)
    {
        printf("Needs m, n and type as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    type = atoi(argv[3]);
    return 0;
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;
    if(type == 0)
    {
        //double precision
        double *dA;
        hipMalloc(&dA, sizeof(double)*m*n);
        int lwork;
        hipsolverHandle_t cusolver_handle ;
        hipsolverDnCreate(&cusolver_handle);
        hipsolverDnDgeqrf_bufferSize(cusolver_handle,
                      m, n, dA, m, &lwork );
        printf("lwork = %d\n", lwork);
        double *work;
        hipMalloc(&work, sizeof(double)*lwork);
        double *tau;
        hipMalloc(&tau, sizeof(double)*m);
        int *info;
        hipMalloc(&info, sizeof(int));
        startTimer();
        hipsolverStatus_t status = hipsolverDnDgeqrf(cusolver_handle,
           m, n, dA, m, tau, work, lwork, info);
        ms = stopTimer();
        printf("cuSOVLER DGEQRF size %dx%d takes %lf ms, tflops is %lf, status = %d\n",m, n, ms, 2.0*n*n*( m -1.0/3.0*n )/(ms*1e9), status);
    }
    else if(type == 1)
    {
        float *dA;
        hipMalloc(&dA, sizeof(float)*m*n);
        int lwork;
        hipsolverHandle_t cusolver_handle ;
        hipsolverDnCreate(&cusolver_handle);
        hipsolverDnSgeqrf_bufferSize(cusolver_handle,
                      m, n, dA, m, &lwork );
        printf("lwork = %d\n", lwork);
        float *work;
        hipMalloc(&work, sizeof(float)*lwork);
        float *tau;
        hipMalloc(&tau, sizeof(float)*m);
        int *info;
        hipMalloc(&info, sizeof(int));
        startTimer();
        hipsolverStatus_t status = hipsolverDnSgeqrf(cusolver_handle,
           m, n, dA, m, tau, work, lwork, info);
        ms = stopTimer();
        printf("cuSOVLER SGEQRF size %dx%d takes %lf ms, tflops is %lf, status = %d\n",m, n, ms, 2.0*n*n*( m -1.0/3.0*n )/(ms*1e9), status);
    }
    else if(type == 2)
    {
        double *dA;
        hipMalloc(&dA, sizeof(double)*m*n);
        int lwork;
        const int blks = m/n;
        hipsolverHandle_t cusolver_handle[blks];
        hipStream_t stream[blks];
        int i = 0;
        for(i = 0; i < m/n; i++)
        {
            hipsolverDnCreate(&cusolver_handle[i]);
            hipStreamCreate(&stream[i]);
            hipsolverSetStream(cusolver_handle[i], stream[i]);
        }
        
        hipsolverDnDgeqrf_bufferSize(cusolver_handle[0],
                      n, n, dA, m, &lwork );
        printf("lwork = %d\n", lwork);
        double *work[m/n];
        for(i = 0; i < m/n; i++)
        {
            hipMalloc(&work[i], sizeof(double)*lwork);
        }
        double *tau[m/n];
        for(i = 0; i < m/n; i++)
        {
            hipMalloc(&tau[i], sizeof(double)*n);
        }
        int *info[m/n];
        for(i = 0; i < m/n; i++)
        {
            hipMalloc(&info[i], sizeof(int));
        }
        startTimer();
        for(i = 0; i < m/n; i++)
        {
            hipsolverStatus_t status = hipsolverDnDgeqrf(cusolver_handle[i],
                n, n, &dA[i*n], m, tau[i], work[i], lwork, info[i]);
        }
        ms = stopTimer();
        printf("cuSOVLER DGEQRF size %dx%d takes %lf ms, tflops is %lf\n",m, n, ms, 2.0*n*n*( m -1.0/3.0*n )/(ms*1e9));
    }
}