#include "hip/hip_runtime.h"
#include "TensorBLAS.h"

long int m, n, nb;
float ms = 0;
bool check = true;
float kernel_ms = 0;
float y_ms = 0;
float dtrsm_ms = 0;
float gemm_ms = 0;

// 确认输入参数，不涉及数据类型
int parseArguments(int argc, char *argv[])
{
    if (argc < 4)
    {
        printf("Needs m, n and nb as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    nb = atoi(argv[3]);
    return 0;
}

// 归并wrap内所有线程的值，值是float的
__inline__ __device__ float warpAllReduceSum(float val)
{
    for (int mask = warpSize / 2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xffffffff, val, mask);
    return val;
}

// 生成单精度的随机矩阵
void generateUniformMatrix(float *dA, long int m, long int n)
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    int seed = 3000;
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateUniform(gen, dA, long(m * n));
}

// 根据论文应该传入M=256行？
// 后续调用的传入: M*N=128*32, blockdim: 32*16
// 此处M和N变量也可以直接作为参数传入给函数，在本程序里效果一样,写成模板可以额外在编译时指定块大小
// 以线程为单位，M：线程块的行数；N：线程块的列数
// 一个线程块处理一个局部矩阵
// 输入时：AA中存放的是待进行QR分解的矩阵，是m&n大小的大矩阵
// 输出时：AA存放的是Q矩阵，RR中存放的是R矩阵
// m,n:总体矩阵的维度
template <long int M, long int N>
__global__ void hou_kernel3(long int m, long int n, float *AA, long int lda, float *RR, long int ldr)
{

    // 计算每个线程块的行数 mm：即当前线程在整个矩阵中需要处理的行数
    // 当前线程所在线程块*M
    long int mm = m - blockIdx.x * M; // 求需要处理的总行数（即去掉已经处理的前blockIdx.x*M行，当前所在的线程块x维度是blockIdx.x，每个线程块的行数为M）
    mm = (mm < M) ? mm : M;           // 实际上是为了计算最后多出的不能被M整除的行数，在此之前就是每个线程处理M行

    if (mm <= 0)
        return;

    const long int mnmin = (mm < n) ? mm : n; 

    float *A = &AA[blockIdx.x * M];   
    float *R = &RR[blockIdx.x * N];    
    __shared__ float As[M * N], Rs[N]; // 定义共享内存，As 用于存储线程块内的局部矩阵的一部分，而 Rs 用于存储一些中间计算的结果
    const long int ldas = M;           // 局部矩阵 As 的列数ldrs = N

    //    float acc0, acc1, acc2, acc3, acc4,acc5, acc6, acc7;
    float acc[8];
    const long int i = threadIdx.x, j = threadIdx.y; // i和j是当前线程在线程块内的索引号

    // #define R07(OP) {OP(0);OP(1);OP(2);OP(3);OP(4);OP(5);OP(6);OP(7);}
    // #define M1(it) if(threadIdx.x+it*32<mm) As[threadIdx.x+it*32+threadIdx.y*ldas] = A[threadIdx.x+it*32+threadIdx.y*lda]

    // #pragma unroll 2
    // 本循环用于将需要处理的分块后的小矩阵先存入共享内存中，降低后续读写成本
    // 此处是在核函数内，一个线程只处理16个元素，但是并行之后可以所有都处理掉
    // 一个线程块有128*32个线程，32*16*16=256*32个矩阵元素
    for (int k = 0; k < 8; k++)
    {
        // 32的来源：blockdim: 32*16，也就是i最大为31，j最大为15
        if (i + k * 32 < mm)
            // i:threadIdx.x，当前线程在块内的索引，第x行
            // k * 32：循环8次，当前是第k次循环
            // j * ldas：j是块内第j列，j * ldas代表按照列优先来编号，前j列（从0开始下标）的所有线程，每列有M行
            // ldas 和 lda 分别表示局部矩阵 As 和全局矩阵子矩阵 A （对应blockIdx.x，即当前线程块所在的那一行的所有线程块组成的子矩阵）在内存中的行跨度
            As[i + k * 32 + j * ldas] = A[i + k * 32 + j * lda]; // As中第i+k*32行第j列的元素
        if (i + k * 32 < mm)
            As[i + k * 32 + (j + 16) * ldas] = A[i + k * 32 + (j + 16) * lda]; // As中第i+k*32行第j+16列的元素
    }

    __syncthreads();

    // 使用循环计算Householder反射，并更新局部矩阵 As。
    // 使用 warpAllReduceSum 函数在warp内进行归约求和操作。
    for (int k = 0; k < mnmin; k++)
    {
        // reference: house_gen.m and house_qr from Cleve Moler blog.
        float nu = 0;
        if (threadIdx.y == k % 16)
        { // threadIdx.y is the warpId; each warp takes two columns

#pragma unroll
            for (int it = 0; it < 8; it++)
            {
    
                (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k) ? (acc[it] = As[threadIdx.x + it * 32 + k * ldas] *
                                                                                        As[threadIdx.x + it * 32 + k * ldas])
                                                                           : acc[it] = 0;
            }
            nu = (acc[0] + acc[1]) + (acc[2] + acc[3]) + (acc[4] + acc[5]) + (acc[6] + acc[7]);
            float normxsqr = (warpAllReduceSum(nu)); 
            float normx = sqrt(normxsqr);            

            float scale = 1.0 / normx; // scale = 1/(v^T*v)

#pragma unroll
            // 对下三角的每个元素执行*scale，即*1/(v^T*v)
            for (int it = 0; it < 8; it++)
            {
                if (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k)
                    As[threadIdx.x + it * 32 + k * ldas] *= scale;
            }
            __syncwarp();
            // k代表列，当x等于k时意味着处理对角线上的元素
            if (threadIdx.x == k)
            {
                float u1 = As[k + k * ldas];
                As[k + k * ldas] += (u1 >= 0) ? 1 : -1;
                Rs[k] = (u1 >= 0) ? -normx : normx;
            }
            __syncwarp();
            // 更新scale为新对角线元素绝对值开根号的倒数
            scale = 1.0 / sqrt(abs(As[k + k * ldas]));
#pragma unroll
            for (int it = 0; it < 8; it++)
            {
                if (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k)
                    As[threadIdx.x + it * 32 + k * ldas] *= scale;
            }

            __syncwarp();
        }
        __syncthreads();

        if (threadIdx.y > k)
        {
            float uxl = 0;
#pragma unroll
            for (int it = 0; it < 8; it++)
            {
                (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k) ? acc[it] = As[threadIdx.x + it * 32 + threadIdx.y * ldas] * As[threadIdx.x + it * 32 + k * ldas] : acc[it] = 0;
            }
            uxl = (acc[0] + acc[1]) + (acc[2] + acc[3]) + (acc[4] + acc[5]) + (acc[6] + acc[7]);
            float ux = warpAllReduceSum(uxl);

#pragma unroll
            for (int it = 0; it < 8; it++)
            {
                if (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k)
                    As[threadIdx.x + it * 32 + threadIdx.y * ldas] -= ux * As[threadIdx.x + it * 32 + k * ldas];
            }
        }
        if (16 + threadIdx.y > k)
        {
            float uxl = 0;
#pragma unroll
            for (int it = 0; it < 8; it++)
            {
                (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k) ? acc[it] = As[threadIdx.x + it * 32 + (16 + threadIdx.y) * ldas] * As[threadIdx.x + it * 32 + k * ldas] : acc[it] = 0;
            }
            uxl = (acc[0] + acc[1]) + (acc[2] + acc[3]) + (acc[4] + acc[5]) + (acc[6] + acc[7]);
            float ux = warpAllReduceSum(uxl);

#pragma unroll
            for (int it = 0; it < 8; it++)
            {
                if (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k)
                    As[threadIdx.x + it * 32 + (16 + threadIdx.y) * ldas] -= ux * As[threadIdx.x + it * 32 + k * ldas];
            }
        }
    }

    __syncthreads();

    // write to R
#pragma unroll
    for (int it = 0; it < 2; it++)
    {
        int j = it * 16 + threadIdx.y;
        int i = threadIdx.x;
        if (i == j)
            // 存放对角线元素
            R[i + i * ldr] = Rs[i];
        // 存放上三角
        else if (i < j)
        {
            R[i + j * ldr] = As[i + j * ldas];
            As[i + j * ldas] = 0;
        }
        // n是传入参数，矩阵的总列数
        else if (i < n)
        {
            R[i + j * ldr] = 0;
        }
    }

    // compute explict Q from Householder reflectors
    double Q[8 * 2];
#pragma unroll
    // Q的对角线元素全部置1，其他为0
    for (int k = 0; k < 8; k++)
    {
        Q[k] = 0;
        Q[k + 8] = 0;
    }
    if (i == j)
        Q[0] = 1.0;
    if (i == j + 16)
        Q[8] = 1.0;

    for (int k = mnmin - 1; k >= 0; k--)
    {
        float acc[8];
        if (threadIdx.y >= k)
        {
            float accum = 0;
            for (int l = 0; l < 8; l++)
                accum += As[i + l * 32 + k * ldas] * Q[l];
            float vq = warpAllReduceSum(accum);

            for (int l = 0; l < 8; l++)
                if (i + 32 * l < mm)
                    Q[l] -= vq * (As[i + 32 * l + k * ldas]);
        }
        if (threadIdx.y + 16 >= k)
        {
            float accum = 0;
            for (int l = 0; l < 8; l++)
                accum += As[i + l * 32 + k * ldas] * Q[l + 8];

            float vq = warpAllReduceSum(accum);
            for (int l = 0; l < 8; l++)
                if (i + 32 * l < mm)
                    Q[l + 8] -= vq * (As[i + 32 * l + k * ldas]);
        }
    }

// 将最终得到的Q矩阵存回A矩阵
#pragma unroll
    for (int k = 0; k < 8; k++)
    {
        if (i + k * 32 < mm)
            A[i + k * 32 + j * lda] = Q[k];
        if (i + k * 32 < mm)
            A[i + k * 32 + (j + 16) * lda] = Q[k + 8];
    }
}

// 由panelQR传入M=128,N=32
template <long int M, long int N>
void hou_caqr_panel(hipblasHandle_t handle, long int m, long int n, float *A, long int lda, float *R, long int ldr, float *work)
{
    // 声明线程块布局，x轴32个线程，y轴16个线程
    dim3 blockdim(32, 16);
    if (m <= M)
    {
        // 传给hou_kernel3也是M=128，N=32
        // 启用1个grid，blockdim个线程块
        hou_kernel3<M, N><<<1, blockdim>>>(m, n, A, lda, R, ldr);
        return;
    }
    if ((m - m / M * M) % N != 0)
    {
        printf("Error: m must be i*%d + j*%d\n", M, N);
    }
    long int NB = (m + M - 1) / M;
    long int ldwork = NB * N;
    long int mm = NB * N;
    hou_kernel3<M, N><<<NB, blockdim>>>(m, n, A, lda, work, ldwork);

    hou_caqr_panel<M, N>(handle, mm, n, work, ldwork, R, ldr, work + ldwork * n);
    float done = 1.0, dzero = 0.0;
    auto status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                            M, N, N,
                                            &done, A, lda, M,
                                            work, ldwork, N,
                                            &dzero, A, lda, M,
                                            m / M);

    mm = m % M;
    if (mm > 0)
    {
        auto status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                  mm, N, N, &done, &A[m / M * M], lda, &work[m / M * N], ldwork,
                                  &dzero, &A[m / M * M], lda);
    }
}

// I - A的结果存进w工作区，划成一个独立的处理矩阵
__global__ void minusEye(long int m, long int n, float *a, long int lda, float *w, long ldw)
{
    long int i = threadIdx.x + blockDim.x * blockIdx.x;
    long int j = threadIdx.y + blockDim.y * blockIdx.y;
    // 条件 means 在矩阵合法索引范围内
    if (i < m && j < n)
    {
        // 求I - A，对角线上元素用1减，非对角线用0减
        if (i == j)
        {
            a[i + j * lda] = 1.0 - a[i + j * lda];
            w[i + j * ldw] = a[i + j * lda];
        }
        else
        {
            a[i + j * lda] = 0.0 - a[i + j * lda];
            w[i + j * ldw] = a[i + j * lda];
        }
    }
}

// 重构生成Y
// reconstructY的参数*U起到了什么作用？
// 输入dA是原矩阵，L和U都存放在dA指向的原地址空间内，其中L的对角线元素是1，dA中的对角线元素是U的对角线元素
void reconstructY(hipsolverHandle_t cusolver_handle, hipblasHandle_t cublas_handle, long m, long n, float *dA, long lda, float *U, float *work, int *info)
{
    // 用于计算单精度矩阵的LU分解
    // 假设P*A=L*U，P是置换矩阵，A是被处理矩阵
    // 此处P为NULL，即不执行轴转，直接A=L*U，输入A，work（工作空间），得到L和U
    // info用于指示是否出错，以及第几个参数出错
    hipsolverDnSgetrf(cusolver_handle, m, n, dA, lda,
                     work, NULL, info);
}

// get U from LU factorization
// U是上三角矩阵，直接取a中的上半
__global__ void getU(int m, int n, float *a, int lda, float *u, int ldu)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n)
    {
        if (i > j)
            u[i + j * ldu] = 0;
        else
            u[i + j * ldu] = a[i + j * lda];
    }
}

// get L from LU factorization
// 把a的上半全部置0，对角线置1，下半就是householder向量
__global__ void getL(int m, int n, float *a, int lda)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n)
    {
        if (i < j)
            a[i + j * lda] = 0;
        else if (i == j)
            a[i + j * lda] = 1;
    }
}

// 在GPU上进行矩阵数组的拷贝，从dA拷贝到dB
__global__ void deviceCopy(long m, long n, float *dB, long ldb, float *dA, long lda)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n)
    {
        dB[i + j * ldb] = dA[i + j * lda];
    }
}

// 把矩阵I变成全0
__global__ void setZero(long m, long n, float *I, long ldi)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n)
    {
        I[i + j * ldi] = 0.0;
    }
}

void panelQR(hipsolverHandle_t cusolver_handle, hipblasHandle_t cublas_handle, long m, long n, float *A, long lda, float *W, long ldw, float *R, long ldr, float *work, int *info)
{
    // 切到最小单位（宽度不超过32）
    if (n <= 32)
    {
        // 中间输出方便纠错
        hipError_t err = hipGetLastError();
        if (err != hipSuccess)
        {
            printf("CUDA error: %s\n", hipGetErrorString(err));
            return;
        }

        startTimer();
        hou_caqr_panel<128, 32>(cublas_handle, m, n, A, lda, R, ldr, work);
        kernel_ms += stopTimer();

        // 设置第一个网格的布局，每个线程块为32*32，网格有多少个块取决于矩阵的大小
        dim3 gridDim((m + 31) / 32, (n + 31) / 32);
        dim3 blockDim(32, 32);

        // W = I - *A = I - Q
        minusEye<<<gridDim, blockDim>>>(m, n, A, lda, W, ldw);
        // deviceCopy<<<gridDim,blockDim>>>( m, n, A, lda, W, ldw );

        // y_ms指在最底层重构Y，即householder vectors的时间
        // 每个时间用+=是因为会执行多次，最后得到总的
        startTimer();
        // means 对I-Q进行LU分解
        reconstructY(cusolver_handle, cublas_handle, m, n, A, lda, work, work + n * n, info);
        y_ms += stopTimer();
        getL<<<gridDim, blockDim>>>(n, n, A, lda);

        float done = 1.0;
        startTimer();
        // 调用cublas库的函数，求解right-rand三角形线性方程组？
        // 此函数可以调用64位int型，means用到了TensorCore？
        // 此时的A是分解后得到的 ，W是 ?
        hipblasStrsm(cublas_handle,
                    HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER,
                    HIPBLAS_OP_T, HIPBLAS_DIAG_UNIT,
                    m, n,
                    &done,
                    A, lda,
                    W, ldw);
        dtrsm_ms += stopTimer();

        return;
    }

    // 递归是把一个宽矩阵切成一个高瘦矩阵，瘦定义为列数不超过32
    panelQR(cusolver_handle, cublas_handle, m, n / 2, A, lda, W, ldw, R, ldr, work, info);

    float done = 1.0, dzero = 0.0, dnegone = -1.0;
    // 计算拖尾矩阵更新的矩阵乘法时间？
    startTimer();
    hipblasSgemm(cublas_handle,
                HIPBLAS_OP_T, HIPBLAS_OP_N,
                n / 2, n / 2, m,
                &done,
                W, ldw,
                A + lda / 2 * n, lda,
                &dzero,
                work, n / 2);
    hipblasSgemm(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n / 2, n / 2,
                &dnegone,
                A, lda,
                work, n / 2,
                &done,
                A + lda / 2 * n, lda);
    gemm_ms += stopTimer();

    dim3 gridDim1((n / 2 + 31) / 32, (n / 2 + 31) / 32);
    dim3 blockDim1(32, 32);
    deviceCopy<<<gridDim1, blockDim1>>>(n / 2, n / 2, R + ldr / 2 * n, ldr, A + lda / 2 * n, lda);
    setZero<<<gridDim1, blockDim1>>>(n / 2, n / 2, A + lda / 2 * n, lda);
    panelQR(cusolver_handle, cublas_handle, m - n / 2, n / 2, A + lda / 2 * n + n / 2, lda, W + ldw / 2 * n + n / 2, ldw, R + n / 2 * ldr + n / 2, ldr, work, info);
    // printf("hrere\n");
    // printMatrixDeviceBlock("R--.csv",n/2,n/2,R+n/2*ldr+n/2,n);

    startTimer();
    hipblasSgemm(cublas_handle,
                HIPBLAS_OP_T, HIPBLAS_OP_N,
                n / 2, n / 2, m,
                &done,
                A, lda,
                W + ldw / 2 * n, ldw,
                &dzero,
                work, n / 2);
    hipblasSgemm(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n / 2, n / 2,
                &dnegone,
                W, ldw,
                work, n / 2,
                &done,
                W + ldw / 2 * n, ldw);
    gemm_ms += stopTimer();
    return;
}

// 把da中的元素拷贝到db中进行存储，再将da全部清空置零
__global__ void copyAndClear(long int m, long int n, float *da, int lda, float *db, int ldb)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n)
    {
        db[i + j * ldb] = da[i + j * lda];
        da[i + j * lda] = 0.0;
    }
}

// 调用hipblasSnrm2函数计算一个单精度矩阵的二范数，即矩阵中所有元素平方和开根号的值
float snorm(long int m, long int n, float *dA)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float sn;
    int incx = 1;
    hipblasSnrm2(handle, m * n, dA, incx, &sn);
    hipblasDestroy(handle);
    return sn;
}

// 分别求原本的A矩阵的二范数，和分解得到的QR相乘得到的A'的二范数，并进行误差对比
void checkResult(long int m, long int n, float *A, long int lda, float *Q, long int ldq, float *R, int ldr)
{
    float normA = snorm(m, n, A);
    float alpha = 1.0;
    float beta = -1.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // printMatrixDeviceBlock("oriA.csv",m,n,A,lda);
    // printMatrixDeviceBlock("QQ.csv",m,n,Q,ldq);
    // printMatrixDeviceBlock("RR.csv",n,n,R,ldr);
    startTimer();
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                m, n, n, &alpha, Q, ldq, R, ldr,
                &beta, A, lda);

    float ms = stopTimer();
    printf("SGEMM m*n*k %d*%d*%d takes %.0f (ms), exec rate %.0f TFLOPS\n",
           m, n, n, ms, 2.0 * m * n * n / (ms * 1e9));
    // printMatrixDeviceBlock("res.csv",m,n,A,lda);
    double normRes = snorm(m, n, A);
    printf("Backward error: ||A-QR||/(||A||) = %.6e\n", normRes / normA);
}

void dorgqr(int m, int n, float *W, int ldw, float *Y, int ldy, float *work)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float sone = 1.0;
    float snegone = -1.0;

    // 线程布局，grid1是线程网格的名字，由处理矩阵的大小横竖来决定，使得一个块固定处理<=32大小的矩阵
    dim3 grid1((m + 1) / 32, (n + 1) / 32);
    // 结合grid，一个线程块就是32*32个线程，每个线程处理一个元素
    dim3 block1(32, 32);
    setEye<<<grid1, block1>>>(m, n, work, m);
    // hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,m,n,n,
    //     &snegone,W,HIP_R_32F, ldw, Y, HIP_R_32F, ldy,
    //     &sone, work, HIP_R_32F, m, HIP_R_32F,
    //     HIPBLAS_GEMM_DEFAULT
    // );
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                m, n, n, &snegone, W, ldw, Y, ldy,
                &sone, work, m);
    hipblasDestroy(handle);
    deviceCopy<<<grid1, block1>>>(m, n, W, ldw, work, m);
}

int main(int argc, char *argv[])
{
    if (parseArguments(argc, argv) == -1)
        return 0;

    float *A;
    hipMalloc(&A, sizeof(float) * m * n);
    float *work;
    hipMalloc(&work, sizeof(float) * m * n);
    float *R;
    hipMalloc(&R, sizeof(float) * n * n);
    float *W;
    hipMalloc(&W, sizeof(float) * m * n);
    float *oriA;

    int *info;
    hipMalloc(&info, sizeof(int));

    hipsolverHandle_t cusolver_handle;
    hipblasHandle_t cublas_handle;
    hipsolverDnCreate(&cusolver_handle);
    hipblasCreate(&cublas_handle);

    generateUniformMatrix(A, m, n);

    if (check)
    {
        hipMalloc(&oriA, sizeof(float) * m * n);
        hipMemcpy(oriA, A, sizeof(float) * m * n, hipMemcpyDeviceToDevice);
    }

    float done = 1.0, dzero = 0.0, dnegone = -1.0;

    for (int i = 0; i < n; i += nb) // nb是最小单位矩阵列数
    {

        panelQR(cusolver_handle, cublas_handle, m - i, nb, A + i * m + i, m, W + i * m + i, m, R + i * n + i, n, work, info);

        if (n - i > nb)
        {
            startTimer();
            hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_T, HIPBLAS_OP_N,
                        nb, n - i - nb, m - i,
                        &done,
                        W + i * m + i, m,
                        A + (i + nb) * m + i, m,
                        &dzero,
                        work, nb);

            hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        m - i, n - i - nb, nb,
                        &dnegone,
                        A + i * m + i, m,
                        work, nb,
                        &done,
                        A + (i + nb) * m + i, m);
            gemm_ms += stopTimer();
            dim3 grid((nb + 1) / 32, (n - i - nb + 1) / 32);
            dim3 block(32, 32);
            copyAndClear<<<grid, block>>>(nb, n - i - nb, A + (i + nb) * m + i, m, R + (i + nb) * n + i, n);
        }

        if (i != 0)
        {
            hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_T, HIPBLAS_OP_N,
                        i, nb, m,
                        &done,
                        A, m,
                        W + i * m, m,
                        &dzero,
                        work, i);

            hipblasSgemm(cublas_handle,
                        HIPBLAS_OP_N, HIPBLAS_OP_N,
                        m, nb, i,
                        &dnegone,
                        W, m,
                        work, i,
                        &done,
                        W + i * m, m);
        }
    }

    ms = kernel_ms + y_ms + dtrsm_ms + gemm_ms;
    printf("kernel: %fms, construct_y: %fms, dtrsm_ms: %fms, gemm_ms: %fms\n", kernel_ms, y_ms, dtrsm_ms, gemm_ms);
    printf("tc_dgeqrf size %dx%d takes %lf ms, tflops is %lf\n", m, n, ms, 2.0 * n * n * (m - 1.0 / 3.0 * n) / (ms * 1e9));

    if (check)
    {
        dorgqr(m, n, W, m, A, m, work);
        checkResult(m, n, oriA, m, W, m, R, n);
    }
}
