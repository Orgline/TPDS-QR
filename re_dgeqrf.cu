#include "hip/hip_runtime.h"
#include "TensorBLAS.h"

long int m, n, nb;

float ms = 0;
bool check=true;
double done = 1.0;
double dzero = 0.0;
double dnegone = -1.0;
double *oriA;


int parseArguments(int argc,char *argv[])
{
    if(argc < 4)
    {
        printf("Needs m, n and nb as inputs\n");
        return -1;
    }
    m = atoi(argv[1]);
    n = atoi(argv[2]);
    nb = atoi(argv[3]);
    return 0;
}

__inline__ __device__ double warpAllReduceSum(double val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xffffffff, val, mask);
    return val;
}

// launch parameters: M*N=256*32, blockdim: 32*16
template<long int M, long int N>
__global__ void hou_kernel3(long int m, long int n, double *AA, long int lda, double *RR, long int ldr )
{

    long int mm = m - blockIdx.x*M; // TB local number of rows
    mm = (mm < M) ? mm : M;

    if (mm <= 0) return;

    const long int mnmin = (mm<n) ? mm : n;

    double *A = &AA[blockIdx.x*M];
    double *R = &RR[blockIdx.x*N];
    __shared__ double As[M*N], Rs[N];
    const long int ldas = M/*, ldrs = N*/;

//    float acc0, acc1, acc2, acc3, acc4,acc5, acc6, acc7;
    double acc[8];
    const long int i=threadIdx.x, j=threadIdx.y;

//#define R07(OP) {OP(0);OP(1);OP(2);OP(3);OP(4);OP(5);OP(6);OP(7);}
//#define M1(it) if(threadIdx.x+it*32<mm) As[threadIdx.x+it*32+threadIdx.y*ldas] = A[threadIdx.x+it*32+threadIdx.y*lda]

//#pragma unroll 2
    for (int k=0; k<8; k++) {
        // FIXME: What if n < 32?
        if(i+k*32<mm) As[i+k*32+j*ldas] = A[i+k*32+j*lda];
        if(i+k*32<mm) As[i+k*32+(j+16)*ldas] = A[i+k*32+(j+16)*lda];
    }

    __syncthreads();

    for (int k=0; k<mnmin; k++) {
        // reference: house_gen.m and house_qr from Cleve Moler blog.
        double nu = 0;

        if(threadIdx.y==k%16) { // threadIdx.y is the warpId; each warp takes two columns
#pragma unroll
            for(int it=0; it<8; it++) {
                (threadIdx.x + it * 32 < mm && threadIdx.x + it * 32 >= k) ?
                (acc[it] = As[threadIdx.x + it * 32 + k * ldas] *
                           As[threadIdx.x + it * 32 + k * ldas]) :
                 acc[it] = 0;
            }
            nu = (acc[0] + acc[1]) + (acc[2] + acc[3]) + (acc[4] + acc[5]) + (acc[6] + acc[7]);

            double normxsqr = (warpAllReduceSum(nu));
            double normx = sqrt(normxsqr);

            double scale = 1.0/normx;

#pragma unroll
            for(int it=0; it<8; it++) {
                if(threadIdx.x+it*32<mm&&threadIdx.x+it*32>=k)
                    As[threadIdx.x+it*32+k*ldas] *= scale;
            }
            __syncwarp();
            if(threadIdx.x==k) {
                double u1 = As[k+k*ldas];

                As[k+k*ldas] += (u1>=0) ? 1 : -1;
                Rs[k] = (u1>=0)? -normx :normx;
            }
            __syncwarp();
            scale = 1.0/sqrt(abs(As[k+k*ldas]));
#pragma unroll
            for(int it=0; it<8; it++) {
                if(threadIdx.x+it*32<mm&&threadIdx.x+it*32>=k)
                    As[threadIdx.x+it*32+k*ldas] *= scale;
            }

            __syncwarp();
        }
        __syncthreads();
        if(threadIdx.y>k) {
            double uxl = 0;
#pragma unroll
            for(int it=0; it<8; it++) {
                (threadIdx.x+it*32<mm&&threadIdx.x+it*32>=k)?
                acc[it]= As[threadIdx.x+it*32+threadIdx.y*ldas] * As[threadIdx.x+it*32+k*ldas]:
                        acc[it] = 0;
            }
            uxl = (acc[0] + acc[1]) + (acc[2] + acc[3]) + (acc[4] + acc[5]) + (acc[6] + acc[7]);
            double ux = warpAllReduceSum(uxl);

#pragma unroll
            for(int it=0; it<8; it++) {
                if(threadIdx.x+it*32<mm&&threadIdx.x+it*32>=k)
                    As[threadIdx.x+it*32+threadIdx.y*ldas] -= ux * As[threadIdx.x+it*32+k*ldas];
            }
        }
        if(16+threadIdx.y>k) {
            double uxl = 0;
#pragma unroll
            for(int it=0; it<8; it++) {
                (threadIdx.x+it*32<mm&&threadIdx.x+it*32>=k)?
                        acc[it]= As[threadIdx.x+it*32+(16+threadIdx.y)*ldas] * As[threadIdx.x+it*32+k*ldas]:
                        acc[it] = 0;
            }
            uxl = (acc[0] + acc[1]) + (acc[2] + acc[3]) + (acc[4] + acc[5]) + (acc[6] + acc[7]);
            double ux = warpAllReduceSum(uxl);

#pragma unroll
            for(int it=0; it<8; it++) {
                if(threadIdx.x+it*32<mm&&threadIdx.x+it*32>=k)
                    As[threadIdx.x+it*32+(16+threadIdx.y)*ldas] -= ux * As[threadIdx.x+it*32+k*ldas];
            }
        }
    }

    __syncthreads();

    // write to R
#pragma unroll
    for (int it=0; it<2; it++) {
        int j = it*16+threadIdx.y;
        int i = threadIdx.x;
        if (i == j)
            R[i + i * ldr] = Rs[i];
        else if (i < j) {
            R[i + j * ldr] = As[i + j * ldas];
            As[i + j * ldas] = 0;

        } else if (i < n) {
            R[i + j * ldr] = 0;
        }
    }



    // compute explict Q from Householder reflectors
    double Q[8*2];
#pragma unroll
    for (int k=0; k<8; k++) {
        Q[k] = 0;
        Q[k+8] = 0;
    }
    if(i==j) Q[0] = 1.0;
    if(i==j+16) Q[8] = 1.0;

    for (int k=mnmin-1; k>=0; k--) {
        double acc[8];
        if(threadIdx.y>=k) {
            double accum = 0;
            for (int l=0; l<8; l++)
                accum += As[i+l*32+k*ldas] * Q[l];
            double vq = warpAllReduceSum(accum);

            for (int l=0; l<8; l++)
                if (i+32*l<mm) Q[l] -= vq*( As[i+32*l + k*ldas] );

        }
        if(threadIdx.y+16>=k) {
            double accum = 0;
            for (int l=0; l<8; l++)
                accum += As[i+l*32+k*ldas] * Q[l+8];

            double vq = warpAllReduceSum(accum);
            for (int l=0; l<8; l++)
                if (i+32*l<mm) Q[l+8] -= vq*( As[i+32*l + k*ldas] );
        }
    }


#pragma unroll
    for (int k=0; k<8; k++) {
        if (i+k*32<mm) A[i+k*32 + j*lda] = Q[k];
        if (i+k*32<mm) A[i+k*32 + (j+16)*lda] = Q[k+8];
    }


}

template<long int M, long int N>
void hou_caqr_panel(hipblasHandle_t handle, long int m, long int n, double *A, long int lda, double *R, long int ldr, double *work)
{
    dim3 blockdim(32, 16);
    if ( m <= M ) {
        hou_kernel3<M, N><<<1,blockdim>>>(m, n, A, lda, R, ldr);
        return;
    }
    if ( (m-m/M*M)%N != 0) {
        printf("Error: m must be i*%d + j*%d\n", M, N);
    }
    long int NB = (m+M-1)/M;
    long int ldwork = NB*N;
    long int mm = NB*N;
    hou_kernel3<M,N><<<NB,blockdim>>>(m, n, A, lda, work, ldwork);

    hou_caqr_panel<M,N>(handle, mm, n, work, ldwork, R, ldr,  work+ldwork*n );
    double done = 1.0, dzero = 0.0;
    auto status = hipblasDgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              M, N, N,
                              &done, A, lda, M,
                              work, ldwork, N,
                              &dzero, A,lda, M,
                              m/M);
    
    mm = m%M;
    if (mm>0) {
        auto status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    mm, N, N, &done, &A[m/M*M], lda, &work[m/M*N], ldwork,
                    &dzero, &A[m/M*M], lda);
    }

}

void generateUniformMatrix(double *dA,long int m,long int n)
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    int seed = 3000;
	hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandGenerateUniformDouble(gen,dA,long(m*n));
}

double dnorm(long int m, long int n, double* dA)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double sn;
    int incx = 1;
    hipblasDnrm2(handle, m*n, dA, incx, &sn);
    hipblasDestroy(handle);
    return sn;
}

void checkResult(long int m,long int n, double* A, long int lda, double *Q, long int ldq, double *R, int ldr)
{
    double normA = dnorm(m,n,A);
    double alpha = 1.0;
    double beta = -1.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // printMatrixDeviceBlock("oriA.csv",m,n,A,lda);
    // printMatrixDeviceBlock("QQ.csv",m,n,Q,ldq);
    // printMatrixDeviceBlock("RR.csv",n,n,R,ldr);
    startTimer();
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    m, n, n, &alpha, Q, ldq, R, ldr,
                    &beta, A, lda);
    
    float ms = stopTimer();
    printf("DGEMM m*n*k %d*%d*%d takes %.0f (ms), exec rate %.0f TFLOPS\n",
            m, n, n, ms, 2.0*m*n*n/(ms*1e9));
    // printMatrixDeviceBlock("res.csv",m,n,A,lda);
    double normRes = dnorm(m,n,A);
    printf("Backward error: ||A-QR||/(||A||) = %.6e\n",normRes/normA);
}

__global__
void setEye(int m, int n, double *a, int lda)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i<m && j<n) 
    {
        if(i==j)
            a[i+j*lda] = 1.0;
        else
            a[i+j*lda] = 0.0;
	}
}

__global__
void deviceCopy( long m, long n, double *dB, long ldb, double *dA, long lda)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i<m && j<n) {
		dB[i+j*ldb] = dA[i+j*lda];
	}
}

void dorgqr(int m, int n, double* W, int ldw, double* Y, int ldy, double* work)
{
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    double sone = 1.0;
    double snegone = -1.0;

    dim3 grid1( (m+1)/32, (n+1)/32 );
	dim3 block1( 32, 32 );
    setEye<<<grid1,block1>>>(m, n, work, m);
    // hipblasGemmEx(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,m,n,n,
    //     &snegone,W,HIP_R_32F, ldw, Y, HIP_R_32F, ldy,
    //     &sone, work, HIP_R_32F, m, HIP_R_32F,
    //     HIPBLAS_GEMM_DEFAULT
    // );
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                    m, n, n, &snegone, W, ldw, Y, ldy,
                    &sone, work, m);
    hipblasDestroy(handle);
    deviceCopy<<<grid1,block1>>>( m, n, W, ldw, work, m);
}

__global__
void minusEye( long int m, long int n, double *a, long int lda, double *w, long ldw)
{
	long int i = threadIdx.x + blockDim.x * blockIdx.x;
	long int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i < m && j < n) {
		if (i == j) 
		{
            a[i+j*lda] = 1.0 - a[i+j*lda];
            w[i+j*ldw] = a[i+j*lda];
        }
		else
		{
            a[i+j*lda] = 0.0 - a[i+j*lda];
            w[i+j*ldw] = a[i+j*lda];
        }
	}
}

// get U from LU factorization
__global__
void getU(int m, int n, double *a, int lda, double *u, int ldu)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i<m && j<n) 
    {
        if (i>j)
            u[i+j*ldu]  = 0;
        else 
            u[i+j*ldu] = a[i+j*lda];
	}
}

// get L from LU factorization
__global__
void getL(int m, int n, double *a, int lda)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i<m && j<n) 
    {
        if (i<j)
            a[i+j*lda] = 0;
        else if (i==j)
            a[i+j*lda] = 1;
	}
}

void reconstructY(hipsolverHandle_t cusolver_handle, hipblasHandle_t cublas_handle, long m, long n, double* dA, long lda, double *U, double *work, int *info)
{
    hipsolverDnDgetrf(cusolver_handle, m, n, dA, lda,
                     work, NULL, info);

    // dim3 gridDim((n+31)/32,(n+31)/32);
    // dim3 blockDim(32,32);
    // getU<<<gridDim,blockDim>>>(n,n,dA,lda,U,n);
    //getL<<<gridDim, blockDim>>>(n,n,dA,lda);

    // double done = 1.0;
    // hipblasDtrsm(cublas_handle,
    //     HIPBLAS_SIDE_RIGHT,  HIPBLAS_FILL_MODE_UPPER,
    //     HIPBLAS_OP_N,  HIPBLAS_DIAG_NON_UNIT,
    //     m-n, n,
    //     &done,
    //     U, n,
    //     dA+n, lda
    // );
}

__global__
void setZero(long m, long n, double *I, long ldi)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
    if (i < m && j < n) 
    {
        I[i+j*ldi] = 0.0;
    }
}

float kernel_ms = 0;
float y_ms = 0;
float dtrsm_ms = 0;
float gemm_ms = 0;


void panelQR(hipsolverHandle_t cusolver_handle, hipblasHandle_t cublas_handle, long m, long n, double *A, long lda, double *W, long ldw, double *R, long ldr, double *work, int *info)
{
    if(n<=32)
    {
        // hipError_t err = hipGetLastError();
        // if (err != hipSuccess) 
        // {
        //     printf("CUDA error: %s\n", hipGetErrorString(err));
        //     return;
        // }
        startTimer();
        
        hou_caqr_panel<128,32>(cublas_handle, m, n, A, lda, R, ldr, work);
        kernel_ms += stopTimer();

        dim3 gridDim((m+31)/32,(n+31)/32);
        dim3 blockDim(32,32);
        
        // setEye<<<gridDim,blockDim>>>(m,n,W,ldw);
        // sSubstract(handle,m,n,A,lda,W,ldw);
        
        minusEye<<<gridDim, blockDim>>>(m, n, A, lda, W, ldw);
        //deviceCopy<<<gridDim,blockDim>>>( m, n, A, lda, W, ldw );
        
        startTimer();
        reconstructY(cusolver_handle, cublas_handle, m, n, A, lda, work, work+n*n, info);
        y_ms += stopTimer();
        getL<<<gridDim, blockDim>>>(n,n,A,lda);
        
        double done = 1.0;
        startTimer();
        hipblasDtrsm(cublas_handle,
            HIPBLAS_SIDE_RIGHT,  HIPBLAS_FILL_MODE_LOWER,
            HIPBLAS_OP_T,  HIPBLAS_DIAG_UNIT,
            m, n,
            &done,
            A, lda,
            W, ldw
        );
        dtrsm_ms+=stopTimer();
        return;
    }
    panelQR(cusolver_handle, cublas_handle, m, n/2, A, lda, W, ldw, R, ldr, work, info);

    double done = 1.0, dzero = 0.0,dnegone = -1.0;
    startTimer();
    hipblasDgemm(cublas_handle,
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            n/2,n/2,m,
            &done,
            W, ldw,
            A+lda/2*n,lda,
            &dzero,
            work,n/2
        );
    hipblasDgemm(cublas_handle,
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m,n/2,n/2,
            &dnegone,
            A, lda,
            work,n/2,
            &done,
            A+lda/2*n,lda
        );
    gemm_ms+=stopTimer();
    dim3 gridDim1((n/2+31)/32,(n/2+31)/32);
    dim3 blockDim1(32,32);
    deviceCopy<<<gridDim1,blockDim1>>>(n/2, n/2, R+ldr/2*n, ldr, A+lda/2*n, lda);
    setZero<<<gridDim1,blockDim1>>>(n/2,n/2,A+lda/2*n,lda);
    panelQR(cusolver_handle, cublas_handle ,m-n/2, n/2, A+lda/2*n+n/2, lda, W+ldw/2*n+n/2, ldw, R+n/2*ldr+n/2, ldr,work, info);
    // printf("hrere\n");
    // printMatrixDeviceBlock("R--.csv",n/2,n/2,R+n/2*ldr+n/2,n);

    

    
    startTimer();
    hipblasDgemm(cublas_handle,
                HIPBLAS_OP_T, HIPBLAS_OP_N,
                n/2,n/2,m,
                &done,
                A, lda,
                W+ldw/2*n,ldw,
                &dzero,
                work,n/2
    );
    hipblasDgemm(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                m,n/2,n/2,
                &dnegone,
                W, ldw,
                work,n/2,
                &done,
                W+ldw/2*n,ldw
            );
    gemm_ms+=stopTimer();
    return;
}

__global__
void copyAndClear( long int m, long int n, double *da, int lda, double *db, int ldb )
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	if (i<m && j<n) {
        db[i+j*ldb] = da[i+j*lda];
        da[i+j*lda] = 0.0;
	}
}


void RHOUQR(hipsolverHandle_t cusolver_handle, hipblasHandle_t cublas_handle, long currentN, long mt, long nt, double* A, long lda, double* W, long ldw, double* R, long ldr, double* work, int* info)
{

    if (nt <= nb) {
        panelQR(cusolver_handle, cublas_handle, mt, nt, A + (currentN - 1) * m + currentN - 1, m, W + (currentN - 1) * m + currentN - 1, m, R + (currentN - 1) * n + currentN - 1, n, work, info);
        return;
    }
    RHOUQR(cusolver_handle, cublas_handle, currentN, mt, nt/2, A, m, W, m, R, n, work, info);

    startTimer();
    hipblasDgemm(cublas_handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N, 
        nt/2, nt/2, mt,    
        &done,
        W + (currentN - 1) * m + currentN - 1, m,
        A + (currentN + nt/2 - 1) * m + currentN - 1, m,
        &dzero,
        work, n);
    hipblasDgemm(cublas_handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        mt, nt/2, nt/2,
        &dnegone,
        A + (currentN - 1) * m + currentN - 1, m,
        work, n,
        &done,
        A + (currentN + nt/2 - 1) * m + currentN - 1, m);
    gemm_ms += stopTimer();
    //填充R，实际上是填充B，R1和R2在分解过程中就已经完成 （n/2, n/2)
    dim3 grid((n + 1) / 32, (n + 1) / 32);   
    dim3 block(32, 32);
    copyAndClear << <grid, block >> > (nt/2, nt/2, A + (currentN + nt/2 - 1) * m + currentN - 1, m, R + (currentN + nt/2 - 1) * n + currentN - 1, n);
    hipDeviceSynchronize();

    //第二次调用mt和nt就需要改变了
    RHOUQR(cusolver_handle, cublas_handle, currentN + nt/2, mt - nt/2, nt/2, A, m, W, m, R, n, work, info);
    hipblasDgemm(cublas_handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
        nt/2, nt/2, mt,
        &done,
        A + (currentN - 1)* m + currentN - 1, m,         //Y1'存储位置
        W + (currentN + nt/2 - 1) * m + currentN - 1, m,         //W2存储位置
        &dzero,
        work, n);
    hipblasDgemm(cublas_handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        mt, nt/2, nt/2,
        &dnegone,
        W + (currentN - 1) * m + currentN - 1, m,
        work, n,
        &done,
        W + (currentN + nt/2 - 1) * m + currentN - 1, m);
}

int main(int argc,char *argv[])
{
    if(parseArguments(argc, argv)==-1)
        return 0;

    double *A;
    hipMalloc(&A, sizeof(double)*m*n);
    double *work;
    hipMalloc(&work, sizeof(double)*m*n);
    double *R;
    hipMalloc(&R, sizeof(double)*n*n);
    double *W;
    hipMalloc(&W, sizeof(double)*m*n);

    int *info;
    hipMalloc (&info, sizeof(int));

    hipsolverHandle_t cusolver_handle;
    hipblasHandle_t cublas_handle;
    hipsolverDnCreate(&cusolver_handle);
    hipblasCreate(&cublas_handle);

    generateUniformMatrix(A,m,n);
    if(check)
    {
        hipMalloc(&oriA, sizeof(double)*m*n);
        hipMemcpy(oriA, A, sizeof(double)*m*n, hipMemcpyDeviceToDevice);
    }
    RHOUQR(cusolver_handle, cublas_handle, 1, m, n, A, m, W, m, R, n, work, info);
    ms = kernel_ms + y_ms + dtrsm_ms + gemm_ms;
    printf("kernel: %fms, construct_y: %fms, dtrsm_ms: %fms, gemm_ms: %fms\n", kernel_ms, y_ms, dtrsm_ms, gemm_ms);
    printf("tc_dgeqrf size %dx%d takes %lf ms, tflops is %lf\n", m, n, ms, 2.0 * n * n * (m - 1.0 / 3.0 * n) / (ms * 1e9));
    if (check)
    {
        dorgqr(m, n, W, m, A, m, work);
        checkResult(m, n, oriA, m, W, m, R, n);
    }
    hipblasDgemm(cublas_handle,
                HIPBLAS_OP_N, HIPBLAS_OP_T,
                m, m, m,
                &done,
                W, m,
                W, m,
                &dzero,
                work, m);
    printf("正交性结果为%f\n",dnorm(m, m, work) * dnorm(m, m, work));

    
    
}
